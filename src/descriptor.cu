#include "hip/hip_runtime.h"
/**
* This file is part of Cuda accelerated ORB-SLAM project by Filippo Muzzini, Nicola Capodieci, Roberto Cavicchioli and Benjamin Rouxel.
 * Implemented by Filippo Muzzini.
 *
 * Based on ORB-SLAM2 (Raúl Mur-Artal, José M.M. Montiel and Juan D. Tardós) and ORB-SLAM3 (Carlos Campos, Richard Elvira, Juan J. Gómez Rodríguez, José M.M. Montiel and Juan D. Tardós)
 *
 * Project under GPLv3 Licence
*
*/

#include <hip/hip_runtime.h>
#include <iostream>

#include "ORBextractor.h"

#include "descriptor.h"

// __device__ __constant__ int HALF_PATCH_SIZE_GPU;

__device__ inline void comp_descr(const uchar *image, ORB_SLAM3::GpuPoint &pt, cv::Point *pattern, int imageStep) {
        const float factorPI = (float)(CV_PI/180.f);
        const float angle = (float)pt.angle*factorPI;
        const float a = (float)cos(angle), b = (float)sin(angle);

        const uchar* center = &(image[(int)pt.y*imageStep+(int)pt.x]);
        const int step = imageStep;

#define GET_VALUE(idx) \
        center[(int)round(pattern[idx].x*b + pattern[idx].y*a)*step + \
               (int)round(pattern[idx].x*a - pattern[idx].y*b)]

        #pragma unroll
        for (int i = 0; i < 32; ++i, pattern += 16)
        {
            int t0, t1, val;
            t0 = GET_VALUE(0); t1 = GET_VALUE(1);
            val = t0 < t1;
            t0 = GET_VALUE(2); t1 = GET_VALUE(3);
            val |= (t0 < t1) << 1;
            t0 = GET_VALUE(4); t1 = GET_VALUE(5);
            val |= (t0 < t1) << 2;
            t0 = GET_VALUE(6); t1 = GET_VALUE(7);
            val |= (t0 < t1) << 3;
            t0 = GET_VALUE(8); t1 = GET_VALUE(9);
            val |= (t0 < t1) << 4;
            t0 = GET_VALUE(10); t1 = GET_VALUE(11);
            val |= (t0 < t1) << 5;
            t0 = GET_VALUE(12); t1 = GET_VALUE(13);
            val |= (t0 < t1) << 6;
            t0 = GET_VALUE(14); t1 = GET_VALUE(15);
            val |= (t0 < t1) << 7;

            pt.descriptor[i] = (uchar)val;
        }

#undef GET_VALUE
    }    

__global__ void compute_descriptor_kernel(uchar *images, uchar *inputImage, ORB_SLAM3::GpuPoint *pointsTotal, const uint *sizes, cv::Point* pattern, int inputImageStep, int maxLevel, const float *mvScaleFactor, int cols, int rows) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int level = blockIdx.y * blockDim.y + threadIdx.y;
    if (level >= maxLevel)
        return;
    
    const uint n = sizes[level];
    if (index >= n) {
        return;
    }

    ORB_SLAM3::GpuPoint *points = &(pointsTotal[level*cols*rows]);

    const uchar* im[2] = {inputImage, &(images[level*cols*rows])};
    const int imIndex = (level == 0) * 0 + (level != 0) * 1;

    const float scale = mvScaleFactor[level];
    const int new_cols = round(cols * 1/scale);
    const int imageStep = (level == 0) * inputImageStep + (level != 0) * new_cols;

    const uchar *myImagePyrimid = im[imIndex];
    
    comp_descr(myImagePyrimid, points[index], pattern, imageStep);

//    printf("level: %d, index: %d\t%d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d\n", level, index,
//           points[index].descriptor[0], points[index].descriptor[1], points[index].descriptor[2], points[index].descriptor[3], points[index].descriptor[4], points[index].descriptor[5], points[index].descriptor[6], points[index].descriptor[7], points[index].descriptor[8], points[index].descriptor[9], points[index].descriptor[10], points[index].descriptor[11], points[index].descriptor[12], points[index].descriptor[13], points[index].descriptor[14], points[index].descriptor[15], points[index].descriptor[16], points[index].descriptor[17], points[index].descriptor[18], points[index].descriptor[19], points[index].descriptor[20], points[index].descriptor[21], points[index].descriptor[22], points[index].descriptor[23], points[index].descriptor[24], points[index].descriptor[25], points[index].descriptor[26], points[index].descriptor[27], points[index].descriptor[28], points[index].descriptor[29], points[index].descriptor[30], points[index].descriptor[31]);

//    points[index].x *= scale;
//    points[index].y *= scale;

}

void compute_descriptor(uchar *images, uchar *inputImage, ORB_SLAM3::GpuPoint *points, uint *sizes, int maxPointsLevel, cv::Point* pattern, int inputImageStep, int maxLevel, int cols, int rows, float *mvScaleFactor, hipStream_t cudaStream){
    dim3 dg( ceil( (float)maxPointsLevel/128 ), ceil((float)maxLevel/8) );
    dim3 db( 128, 8 );

    compute_descriptor_kernel<<<dg, db, 0, cudaStream>>>(images, inputImage, points, sizes, pattern, inputImageStep, maxLevel, mvScaleFactor, cols, rows);
}
